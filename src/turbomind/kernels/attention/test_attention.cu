#include "hip/hip_runtime.h"
// Copyright (c) OpenMMLab. All rights reserved.

#include "attention.h"
#include "kv_cache.h"
#include "src/turbomind/kernels/attention/reference.h"
#include "src/turbomind/kernels/unfused_attention_kernels.h"
#include "test_utils.h"
#include <cmath>
#include <ios>
#include <iostream>
#include <limits>
#include <thrust/universal_vector.h>

#include <algorithm>
#include <iomanip>
#include <numeric>
#include <random>

using namespace turbomind;

// [S/S, H, S, D] <-> [S/b, H, b, D]
void TestBlocks(const thrust::universal_vector<half>& k_cache,  // [B, H, S, D]
                const thrust::universal_vector<half>& v_cache,  // [B, H, S, D]
                thrust::universal_vector<half>&       blocks,   // block data
                thrust::universal_vector<half*>&      k_ptrs,   // block ptrs
                thrust::universal_vector<half*>&      v_ptrs,
                thrust::universal_vector<int>&        cu_block_cnts,  // cumulative block counts
                const int                             head_num,
                const int                             head_dim,
                const int                             block_seq_len,
                const int                             batch_size)
{
    const int seq_len  = k_cache.size() / (head_dim * head_num * batch_size);
    const int n_blocks = (seq_len + block_seq_len - 1) / block_seq_len;

    const int kHSD = head_num * seq_len * head_dim;

    std::cout << "batch_size = " << batch_size << ", seq_len = " << seq_len << ", block_size = " << block_seq_len
              << ", block_num = " << n_blocks << "\n";

    thrust::universal_vector<half> kv_cache(k_cache.size() * 2);  // [B, 2, H, S, D]

    {  // interleave K/V
        auto k_src = k_cache.begin();
        auto v_src = v_cache.begin();
        auto dst   = kv_cache.begin();
        for (int i = 0; i < batch_size; ++i) {
            dst = thrust::copy_n(k_src, kHSD, dst);
            dst = thrust::copy_n(v_src, kHSD, dst);
            k_src += kHSD;
            v_src += kHSD;
        }
    }

    const int kHsD = head_num * block_seq_len * head_dim;

    // [B, S/s, 2, H, s, D]
    blocks.resize(batch_size * n_blocks * 2 * kHsD);
    thrust::fill(blocks.begin(), blocks.end(), NAN);
    k_ptrs.resize(batch_size * n_blocks + 1);  // +1 padding
    v_ptrs.resize(batch_size * n_blocks + 1);

    std::vector<size_t> idxs(batch_size * n_blocks);
    std::iota(idxs.begin(), idxs.end(), 0);

    std::random_device rd;
    std::mt19937       g(rd());
    std::shuffle(idxs.begin(), idxs.end(), g);

    for (size_t i = 0; i < idxs.size(); ++i) {
        k_ptrs[i] = blocks.data().get() + idxs[i] * 2 * kHsD;
        v_ptrs[i] = k_ptrs[i] + kHsD;
    }

    thrust::universal_vector<int> seq_lens(batch_size);
    thrust::fill(seq_lens.begin(), seq_lens.end(), seq_len);

    std::vector<int> n_blocks_vec(batch_size + 1, n_blocks);
    cu_block_cnts.resize(batch_size + 1);
    std::exclusive_scan(n_blocks_vec.begin(), n_blocks_vec.end(), cu_block_cnts.begin(), 0);

    // [B, 2H, S, D] -> [B, S/s] x [2H, s, D]
    for (int i = 0; i < 1; ++i) {
        ConvertLinearToBlocks((const half*)kv_cache.data().get(),
                              k_ptrs.data().get(),
                              cu_block_cnts.data().get(),
                              seq_lens.data().get(),
                              0,
                              seq_len,
                              block_seq_len,
                              2 * head_num,
                              head_dim,
                              batch_size,
                              0);
    }

    thrust::universal_vector<half> kv_cache_2(kv_cache.size());
    // round trip test
    for (int i = 0; i < 1; ++i) {
        ConvertBlocksToLinear((const half**)k_ptrs.data().get(),
                              kv_cache_2.data().get(),
                              cu_block_cnts.data().get(),
                              seq_lens.data().get(),
                              0,
                              block_seq_len,
                              seq_len,
                              2 * head_num,
                              head_dim,
                              batch_size,
                              0);
    }
    hipDeviceSynchronize();

    if (0) {
        std::cout << ">>> Compare\n";
        Compare(
            kv_cache.data().get(), kv_cache_2.data().get(), head_dim, head_dim, batch_size * 2 * head_num * seq_len);
        std::cout << "<<< Compare\n";
    }
}

int main(int argc, char* argv[])
{
    AttentionParams<half> params{};

    constexpr int kHeadNum = 16;
    // constexpr int kHeadNum     = 1;
    constexpr int kHeadDim   = 128;
    constexpr int KvHeadNum  = kHeadNum;
    constexpr int kBatchSize = 2;
    // constexpr int kBatchSize   = 1;
    constexpr int kInputLen    = 8192;
    constexpr int kSequenceLen = 0;
    // constexpr int kInputLen    = 4096 - 20;
    // constexpr int kSequenceLen = 32 + 16 + 8 + 4;  // force partial tile
    // constexpr int kSequenceLen = 983;
    // constexpr int kInputLen    = 2387;
    // constexpr int kSequenceLen = 72;
    // constexpr int kInputLen    = 98;
    constexpr int kContextLen = kSequenceLen + kInputLen;
    constexpr int kBlockSz    = 64;
    constexpr int kTestIter   = 10;
    constexpr int kMaxSplitK  = 1;

    RNG rng{};

    thrust::universal_vector<half> k_cache(kBatchSize * KvHeadNum * kContextLen * kHeadDim);
    thrust::universal_vector<half> v_cache(kBatchSize * KvHeadNum * kContextLen * kHeadDim);

    thrust::universal_vector<half> qkv(kBatchSize * kInputLen * (kHeadNum + KvHeadNum * 2) * kHeadDim);
    thrust::universal_vector<half> output(kBatchSize * kInputLen * kHeadNum * kHeadDim);

    thrust::universal_vector<bool> finished(kBatchSize);
    thrust::universal_vector<int>  sequence_length(kBatchSize);
    thrust::universal_vector<int>  input_length(kBatchSize);
    thrust::universal_vector<int>  context_length(kBatchSize);
    thrust::universal_vector<int>  cu_seqlens(kBatchSize + 1);

    thrust::universal_vector<float> partial_M(kBatchSize * kHeadNum * kMaxSplitK);
    thrust::universal_vector<float> partial_L(kBatchSize * kHeadNum * kMaxSplitK);
    thrust::universal_vector<float> partial_O(kBatchSize * kHeadNum * kMaxSplitK * kHeadDim);
    thrust::universal_vector<int>   semaphores(kBatchSize * kHeadNum * kMaxSplitK);

    std::fill(semaphores.begin(), semaphores.end(), 0);

    rng.GenerateNormal(qkv.data().get(), qkv.size(), 1.f, 0.f);

    rng.GenerateNormal(k_cache.data().get(), kBatchSize * KvHeadNum * kContextLen * kHeadDim);
    rng.GenerateNormal(v_cache.data().get(), kBatchSize * KvHeadNum * kContextLen * kHeadDim);

    // Set input range to zero
    // (BH, SD)
    hipMemset2DAsync(k_cache.data().get() + kSequenceLen * kHeadDim,
                      sizeof(half) * kContextLen * kHeadDim,
                      0,
                      sizeof(half) * kInputLen * kHeadDim,
                      kBatchSize * KvHeadNum);
    hipMemset2DAsync(v_cache.data().get() + kSequenceLen * kHeadDim,
                      sizeof(half) * kContextLen * kHeadDim,
                      0,
                      sizeof(half) * kInputLen * kHeadDim,
                      kBatchSize * KvHeadNum);

    thrust::universal_vector<half>  blocks;
    thrust::universal_vector<half*> k_ptrs;
    thrust::universal_vector<half*> v_ptrs;
    thrust::universal_vector<int>   cu_block_cnts;

    TestBlocks(k_cache, v_cache, blocks, k_ptrs, v_ptrs, cu_block_cnts, KvHeadNum, kHeadDim, kBlockSz, kBatchSize);

    thrust::universal_vector<half>  k_cache_ref = k_cache;
    thrust::universal_vector<half>  v_cache_ref = v_cache;
    thrust::universal_vector<half>  output_ref  = output;
    thrust::universal_vector<void*> k_cache_ref_ptrs(kBatchSize);
    thrust::universal_vector<void*> v_cache_ref_ptrs(kBatchSize);

    hipDeviceSynchronize();

    for (int i = 0; i <= kBatchSize; ++i) {
        cu_seqlens[i] = i * kInputLen;
    }

    for (int i = 0; i < kBatchSize; ++i) {
        input_length[i]     = kInputLen;
        sequence_length[i]  = kSequenceLen;
        context_length[i]   = kContextLen;
        k_cache_ref_ptrs[i] = k_cache_ref.data().get() + i * k_cache_ref.size() / kBatchSize;
        v_cache_ref_ptrs[i] = v_cache_ref.data().get() + i * v_cache_ref.size() / kBatchSize;
    }

    // getchar();

    params.out    = output_ref.data().get();
    params.q      = qkv.data().get();
    params.k      = params.q + kHeadNum * kHeadDim;
    params.v      = params.k + KvHeadNum * kHeadDim;
    params.stride = (kHeadNum + 2 * KvHeadNum) * kHeadDim;

    params.batch_size    = kBatchSize;
    params.max_input_len = kInputLen;
    params.max_seq_len   = kSequenceLen;
    params.cu_block_cnts = cu_block_cnts.data().get();

    params.k_cache_block_ptrs = (void**)k_ptrs.data().get();
    // params.v_cache_block_ptrs  = (void**)v_ptrs.data().get();
    params.kv_cache_block_size = kBlockSz;

    params.finished       = finished.data().get();
    params.input_length   = input_length.data().get();
    params.context_length = context_length.data().get();
    params.cu_seqlens     = cu_seqlens.data().get();
    // params.layer_offset   = 0;
    // [L, 2, H, s, D]
    params.key_offset = 0;
    params.val_offset = params.key_offset + KvHeadNum * kBlockSz * kHeadDim;

    params.num_heads     = kHeadNum;
    params.num_kv_heads  = KvHeadNum;
    params.size_per_head = kHeadDim;
    params.inv_sqrt_dh   = M_LOG2E / std::sqrt((float)params.size_per_head);

    params.rotary_embedding_dim  = kHeadDim;
    params.rotary_embedding_base = 10000.f;

    params.partial_L = partial_L.data().get();
    params.partial_M = partial_M.data().get();
    params.partial_O = partial_O.data().get();
    params.locks     = semaphores.data().get();

    params.max_split_k = kMaxSplitK;
    params.arch        = 80;

    // thrust::universal_vector<float> qk(kBatchSize * kInputLen * kSequenceLen);
    // thrust::universal_vector<half>  pr(kBatchSize * kInputLen * kSequenceLen);
    // params.qk = qk.data().get();
    // params.pr = pr.data().get();

    Reference<half> reference(Reference<half>::kFLASH_ATTENTION, {});
    reference.Reshape(kInputLen, kContextLen, kHeadNum, kHeadDim, KvHeadNum, kBatchSize);

    for (int i = 0; i < 10; ++i) {
        // mmha_ft_reference(params,
        //                   (half**)k_cache_ref_ptrs.data().get(),
        //                   (half**)v_cache_ref_ptrs.data().get(),
        //                   sequence_length.data().get(),
        //                   kContextLen,
        //                   hipStream_t{});

        reference.Execute(params.out, k_cache_ref.data().get(), v_cache_ref.data().get(), qkv.data().get());
    }

    hipDeviceSynchronize();

    if (0) {
        for (int b = 0; b < kBatchSize; ++b) {
            for (int i = 0; i < kInputLen; ++i) {
                auto qk = reference.qk() + b * kInputLen * kContextLen + i * kInputLen;
                for (int j = 0; j < kInputLen; ++j) {
                    std::cout << qk[j] * params.inv_sqrt_dh << " ";
                }
                std::cout << "\n";
            }
            std::cout << "\n";
        }

        std::cout << "\n";

        // for (int i = 0; i < kInputLen; ++i) {
        //     auto pr = unfused_attention.pr() + i * kInputLen;
        //     for (int j = 0; j < kInputLen; ++j) {
        //         std::cout << (float)pr[j] << " ";
        //     }
        //     std::cout << "\n";
        // }
    }

    if (auto err = hipGetLastError(); err != hipSuccess) {
        std::cout << hipGetErrorString(err) << "\n";
        return -1;
    }
    std::cout << "---------------------------------------------------\n";

    params.out = output.data().get();

    std::vector<thrust::universal_vector<half>> outputs;

    for (int i = 0; i < std::max(kTestIter, 1); ++i) {
        invokeProcessKV<half>(params);
        dispatchAttention<half>(params);
        if (auto err = hipGetLastError(); err != hipSuccess) {
            std::cout << hipGetErrorString(err) << "\n";
            return -1;
        }
        if (1) {
            outputs.push_back(output);
        }
    }

    if (1) {
        ConvertBlocksToLinear((const half**)k_ptrs.data().get(),
                              k_cache.data().get(),
                              cu_block_cnts.data().get(),
                              context_length.data().get(),
                              0,
                              kBlockSz,
                              kContextLen,
                              KvHeadNum,
                              kHeadDim,
                              kBatchSize,
                              0);
        ConvertBlocksToLinear((const half**)v_ptrs.data().get(),
                              v_cache.data().get(),
                              cu_block_cnts.data().get(),
                              context_length.data().get(),
                              0,
                              kBlockSz,
                              kContextLen,
                              KvHeadNum,
                              kHeadDim,
                              kBatchSize,
                              0);
    }

    hipDeviceSynchronize();

    if (outputs.size() > 1) {
        std::cout << "Evaluating consistency..." << std::endl;
        for (size_t i = 1; i < outputs.size(); ++i) {
            Compare(outputs[i].data().get(), outputs[0].data().get(), kHeadDim, kHeadDim, kHeadNum);
        }
    }

    std::cout << "---------------------------------------------------\n";

    // [B, S, H, D]
    // Compare(output.data().get(),  //
    //         output_ref.data().get(),
    //         kHeadDim,
    //         kHeadDim,
    //         kBatchSize * kInputLen * kHeadNum,
    //         0);
    Compare(output.data().get(),  //
            output_ref.data().get(),
            kHeadNum * kHeadDim,
            kHeadNum * kHeadDim,
            kBatchSize * kInputLen,
            0);

    // [BH, SD]
    Compare(k_cache.data().get() + kSequenceLen * kHeadDim,
            k_cache_ref.data().get() + kSequenceLen * kHeadDim,
            kContextLen * kHeadDim,
            kInputLen * kHeadDim,
            kBatchSize * KvHeadNum);
    Compare(v_cache.data().get() + kSequenceLen * kHeadDim,
            v_cache_ref.data().get() + kSequenceLen * kHeadDim,
            kContextLen * kHeadDim,
            kInputLen * kHeadDim,
            kBatchSize * KvHeadNum);

    return 0;
}

// if (0) {
//     for (int i = 0; i < kBatchSize; ++i) {
//         std::cout << "Compare Keys " << i << ": ";
//         Compare((half*)params.k_cache_block_ptrs[cu_block_cnts[i]],                 // [H, s, D]
//                 k_cache_ref.data().get() + i * KvHeadNum * kContextLen * kHeadDim,  // [H, S, D]
//                 kHeadDim,
//                 kHeadDim,
//                 KvHeadNum * kContextLen,
//                 1);
//     }
// }